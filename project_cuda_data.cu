//Parallel Programming Final Project (CUDA)
//Team: 22
//ver 1.2		2018/12/16	16:20

#include <iostream>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <vector>
#include <time.h>
#include <hip/hip_runtime.h>


using namespace std;

int NUM_STEPS;
int NUM_DATA;
double 	*C_gpu,
		*P_gpu,
		*C_price_gpu,
		*P_price_gpu;

struct data {
    double K; 
    double S;
    double r;
    double v;
    double T;
	double dt;
	double vdt;
	double u;
	double d;
	double p;
	double* last_C;
	double* last_P;
	double C_price;
	double P_price;
}; 

typedef struct data option_data;

void init(option_data *in){
    in->r = 0.005 ;   // Risk-free rate (5%)
    in->v = 0.3 ;    // Volatility of the underlying (20%)
    in->T = 1.0;   // One year until expiry
	in->dt = in->T/NUM_STEPS;
	in->vdt = in->v*sqrt(in->dt);
	in->u = exp(in->vdt);
	in->d = 1/in->u;
	in->p = (exp(in->r*in->dt)-in->d)/(in->u-in->d);
	in->last_C = new double [NUM_STEPS+1];
	in->last_P = new double [NUM_STEPS+1];
}

//Call Option 認購期權
double CallOption(const double& S,const double& K,const double& vDt,const int& i){
	double d = S * exp(vDt * (2.0 * i - NUM_STEPS)) - K;      
	return max(d,(double)0); 
}

//Put Option 認沽期權
double PutOption(const double& S,const double& K,const double& vDt,const int& i){
	double d = K-S * exp(vDt * (2.0 * i - NUM_STEPS));          
	return max(d,(double)0); 
}

void last_step_price(option_data* temp_data,int NUM_STEPS){
	for(int k = 0; k <=NUM_STEPS; k++){
		double sd = temp_data->S * exp(temp_data->vdt * (2.0 * (NUM_STEPS-k) - NUM_STEPS)) ; 
		
		temp_data->last_C[k] = max(sd-temp_data->K,(double)0);
		temp_data->last_P[k] = max(temp_data->K-sd,(double)0);
	}
	
	return;
}



__global__ void trace_back_gpu(double* C_option, double* P_option, double* C_price, double* P_price, double r, double dt, double p, int NUM_DATA, int NUM_STEPS){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if(x<NUM_DATA){
		for(int t = 0; t < NUM_STEPS; ++t){
			for(int i = 0; i < NUM_STEPS-t; ++i){
				C_option[x*(NUM_STEPS+1)+i] = exp(-r*dt) * (p*C_option[x*(NUM_STEPS+1)+i] + (1-p)*C_option[x*(NUM_STEPS+1)+i+1]);
				P_option[x*(NUM_STEPS+1)+i] = exp(-r*dt) * (p*P_option[x*(NUM_STEPS+1)+i] + (1-p)*P_option[x*(NUM_STEPS+1)+i+1]);
			}
		}			
	}

	C_price[x] = C_option[x*(NUM_STEPS+1)];	
	P_price[x] = P_option[x*(NUM_STEPS+1)];

	return;
}

int main(int argc, char **argv){
	if(argc!=3){
		cout << "Input error!\n";
		cout << "./project <NUM_STEPS> <DATA_FILE>\n";
		return 1;
	}
	sscanf(argv[1],"%d",&NUM_STEPS);
	ifstream infile(argv[2]);
	if(!infile) {
		cout << "Can not open input file!\n";
		return 1;
	}
	
	infile >> NUM_DATA;
	cout << "num_data = " << NUM_DATA << endl;
	cout << "num_step = " << NUM_STEPS << endl;
    
	size_t size = ((NUM_STEPS+1)*NUM_DATA)*sizeof(double);
	vector<option_data*> option_vector;
	
	for(int i = 0; i < NUM_DATA; ++i){
		double K, S; 
		infile >> K >> S;
	    option_data *temp_data = new option_data();
		init(temp_data);
		temp_data->K = K;
		temp_data->S = S;
		
		last_step_price(temp_data,NUM_STEPS);
        option_vector.push_back(temp_data);
	}

	double** C_matrix = new double*[NUM_DATA];
	double** P_matrix = new double*[NUM_DATA];
	double* C_array = new double[(NUM_STEPS+1)*NUM_DATA];
	double* P_array = new double[(NUM_STEPS+1)*NUM_DATA];
	
	for(int i = 0; i < NUM_DATA; ++i){
		C_matrix[i] = option_vector[i]->last_C;
		P_matrix[i] = option_vector[i]->last_P;
		for(int j = 0; j < NUM_STEPS+1; ++j){
			C_array[i*(NUM_STEPS+1)+j] = C_matrix[i][j];
			P_array[i*(NUM_STEPS+1)+j] = P_matrix[i][j];
		}
	}

	double r = option_vector[0]->r;
	double dt = option_vector[0]->dt;
	double p = option_vector[0]->p;
	double C_price[NUM_DATA];
	double P_price[NUM_DATA];


    
	hipMalloc((void**)&C_gpu, size);
	hipMalloc((void**)&P_gpu, size);
	hipMalloc((void**)&C_price_gpu, NUM_DATA*sizeof(double));
	hipMalloc((void**)&P_price_gpu, NUM_DATA*sizeof(double));
	hipMemcpy(C_gpu, C_array, size, hipMemcpyHostToDevice);
	hipMemcpy(P_gpu, P_array, size, hipMemcpyHostToDevice);
	//trace_back_gpu<<<NUM_DATA/32 + 1,32>>>(C_gpu, C_price_gpu, r, dt, p, NUM_DATA, NUM_STEPS);
	trace_back_gpu<<<NUM_DATA/32 + 1,32>>>(C_gpu, P_gpu, C_price_gpu, P_price_gpu, r, dt, p, NUM_DATA, NUM_STEPS);
	hipMemcpy(C_price, C_price_gpu, NUM_DATA*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(P_price, P_price_gpu, NUM_DATA*sizeof(double), hipMemcpyDeviceToHost);
	
	//print
	for(int i =0;i<NUM_DATA;i++){
	    option_data* print_price = option_vector[i];
		printf("Data %d: Call Price: %.5f\tPut Price: %.5f\n", i, C_price[i], P_price[i]);
	}
	
	return 0;
}
